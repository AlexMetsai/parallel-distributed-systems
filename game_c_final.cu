#include "hip/hip_runtime.h"
/* Metsai Aleksandros 7723
 * metsalex@ece.auth.gr
 * 
 * Multiple cells per thread and use of shared memory
 */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define THRESHOLD 0.4


#define CELLS_PER_THREAD 2
#define THREADS_PER_BLOCK (500/CELLS_PER_THREAD)


struct timeval startwtime, endwtime;
double seq_time;


__global__ void game_c (int *newer, int *old, int N)
{
	
  int lsize=THREADS_PER_BLOCK*CELLS_PER_THREAD;
  
  __shared__ int top[THREADS_PER_BLOCK*CELLS_PER_THREAD+2];		//Extended Tables
  __shared__ int mid[THREADS_PER_BLOCK*CELLS_PER_THREAD+2];
  __shared__ int bot[THREADS_PER_BLOCK*CELLS_PER_THREAD+2];
  
  int index = blockIdx.x*blockDim.x*CELLS_PER_THREAD + threadIdx.x*CELLS_PER_THREAD;
  int count;
  int sum=0;
  int i=(int) index/N;
  int j= index%N;
  int lindex = threadIdx.x*CELLS_PER_THREAD +1; //Local Index
  
  for(count=0; count<CELLS_PER_THREAD;count++){
	  
	  
	  if(i==0){
		  
		  //top
		  
		  if(j==0){
			  
			  top[0]= old[N*N-1];
			  mid[0]= old[N-1];
			  bot[0]= old[2*N -1];
			  
			  top[1]= old[N*(N-1)];
			  mid[1]= old[0];
			  bot[1]= old[N];
			  
		  }else if(j==(N-1)){
			  
			  top[lindex+1]= old[N*(N-1)];
			  mid[lindex+1]= old[0];
			  bot[lindex+1]= old[N];
			  
			  top[lindex]= old[N*N -1];
			  mid[lindex]= old[N-1];
			  bot[lindex]= old[2*N -1];
			  
		  }else{
			  if(lindex==1){
				  
				  top[lindex-1]= old[N*(N-1) +(j-1)];
				  mid[lindex-1]= old[j-1];
				  bot[lindex-1]= old[N+(j-1)];
				  
				  top[lindex]= old[N*(N-1) +j];
				  mid[lindex]= old[j];
				  bot[lindex]= old[N+j];
				  
			  }else if(lindex==(lsize)){
				  
				  top[lindex+1]= old[N*(N-1) +(j+1)];
				  mid[lindex+1]= old[j+1];
				  bot[lindex+1]= old[N+ (j+1)];
				  
				  top[lindex]= old[N*(N-1) +j];
				  mid[lindex]= old[j];
				  bot[lindex]= old[N +j];
				  
			  }else{
				  
				  top[lindex]= old[N*(N-1) +j];
				  mid[lindex]= old[j];
				  bot[lindex]= old[N+j];
				  
			  }
		  }
	  }else if(i==(N-1)){
		  
		  //bottom
		  
		  if(j==0){
			  
			  top[0]= old[N*(N-1) -1];
			  mid[0]= old[N*N -1];
			  bot[0]= old[N-1];
			  
			  top[1]= old[N*(N-2)];
			  mid[1]= old[N*(N-1)];
			  bot[1]= old[0];
			  
		  }else if(j==(N-1)){
			  
			  top[lindex+1]= old[N*(N-2)];
			  mid[lindex+1]= old[N*(N-1)];
			  bot[lindex+1]= old[0];
			  
			  top[lindex]= old[N*(N-1) -1];
			  mid[lindex]= old[N*N -1];
			  bot[lindex]= old[N-1];
			  
		  }else{
			  // !!
			  if(lindex==1){
			  
				top[lindex-1]= old[(i-1)*N +(j-1)];
				mid[lindex-1]= old[i*N +(j-1)];
				bot[lindex-1]= old[j-1];
			  
				top[lindex]= old[(i-1)*N +j];
				mid[lindex]= old[i*N +j];
				bot[lindex]= old[j];
			  
			  }else if(lindex==(lsize)){
			  
			   top[lindex+1]= old[(i-1)*N +(j+1)];
			   mid[lindex+1]= old[i*N +(j+1)];
			   bot[lindex+1]= old[(j+1)];
			  
			   top[lindex]= old[(i-1)*N +j];
			   mid[lindex]= old[i*N +j];
			   bot[lindex]= old[j];
			   
		    }else{
		  
				top[lindex]= old[(i-1)*N +j];
				mid[lindex]= old[i*N +j];
				bot[lindex]= old[j];
			}
			  
		 }
	  }else if(j==0){
		  
		  //left
		  
		  top[0]= old[(i-1)*N +(N-1)];
		  mid[0]= old[i*N +(N-1)];
		  bot[0]= old[(i+1)*N +(N-1)];
		  
		  top[1]= old[(i-1)*N];
		  mid[1]= old[i*N];
		  bot[1]= old[(i+1)*N];
		  
	  }else if(j==(N-1)){
		  
		  //right
		  
		  top[lindex+1]= old[(i-1)*N];
		  mid[lindex+1]= old[i*N];
		  bot[lindex+1]= old[(i+1)*N];
		  
		  top[lindex]= old[(i-1)*N +j];
		  mid[lindex]= old[i*N +j];
		  bot[lindex]= old[(i+1)*N +j];
		  
	  }else{
		  
		  //general case
		  
		  if(lindex==1){
			  
			  top[lindex-1]= old[(i-1)*N +(j-1)];
			  mid[lindex-1]= old[i*N +(j-1)];
			  bot[lindex-1]= old[(i+1)*N +(j-1)];
			  
			  top[lindex]= old[(i-1)*N +j];
			  mid[lindex]= old[i*N +j];
			  bot[lindex]= old[(i+1)*N +j];
			  
		  }else if(lindex==(lsize)){
			  
			  top[lindex+1]= old[(i-1)*N +(j+1)];
			  mid[lindex+1]= old[i*N +(j+1)];
			  bot[lindex+1]= old[(i+1)*N +(j+1)];
			  
			  top[lindex]= old[(i-1)*N +j];
			  mid[lindex]= old[i*N +j];
			  bot[lindex]= old[(i+1)*N +j];
		  }else{
		  
			top[lindex]= old[(i-1)*N +j];
			mid[lindex]= old[i*N +j];
			bot[lindex]= old[(i+1)*N +j];
			
		  }
	  }
	  
	  lindex++;
	  j++;
  }
  
  //Restore values
  j=index%N;
  lindex=threadIdx.x*CELLS_PER_THREAD +1;
  
  __syncthreads();
  
  for(count=0; count<CELLS_PER_THREAD; count++){
	  
	  sum= top[lindex-1] +top[lindex]+top[lindex+1]
	  +mid[lindex-1] +mid[lindex+1]
	  +bot[lindex-1] +bot[lindex] +bot[lindex+1];
	  
	  switch(sum){
		  
		  case 3:
		  newer[i*N + j] = 1;
		  break;
		  
		  case 2:
		  newer[i*N + j] = old[i*N + j];
		  break;
		  
		  default:
		  newer[i*N + j] = 0;
	  }
	  
	  lindex++;
	  j++;
  }

}


void read_from_file(int *X, char *filename, int N);
void save_table(int *X, int N);


int main(){
	

	int *table;
	int* newer;
	int* old;
	int *temp;
	
	int blocks, t, N, count;

	
	printf("Set the number of generations\n");
	scanf("%d", &t);
	printf("Set N (table size = NxN)\n");
	scanf("%d", &N);
	int size=N*N*sizeof(int);
	
	/*
	 Insert table here
	 */
	
	
	char filename[20];
	sprintf(filename, "table%dx%d.bin", N, N);
	printf("Reading %dx%d table from file %s\n", N, N, filename);
	table = (int *)malloc(N*N*sizeof(int));
	read_from_file(table, filename, N);  	
	

	printf("This is kernel c\n");
	
	printf("The game will be played for %d generations N=%d\n", t, N);
	
	//!!!Start Timer!!!
	gettimeofday (&startwtime, NULL);
	
	//Allocate space of new and old in device
	hipMalloc(&newer, size);
	hipMalloc(&old, size);
	
	//copy table
	hipMemcpy(old, table, size, hipMemcpyHostToDevice);
	
	blocks=(N*N)/(THREADS_PER_BLOCK*CELLS_PER_THREAD);
	
	//Play game for t generations
	for(count=0;count<t;count++){
		
		game_c<<<blocks, THREADS_PER_BLOCK>>>(newer, old, N);
		hipDeviceSynchronize();
		
		//swap pointers
		temp=old;
		old=newer;
		newer=temp;
	}
	
	//copy back table
	hipMemcpy(table, old, size, hipMemcpyDeviceToHost);
	
	//!!!End Timer!!!
	gettimeofday (&endwtime, NULL);
	  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
		      + endwtime.tv_sec - startwtime.tv_sec);
	  printf("Cuda clock time = %f\n", seq_time);
	
	save_table(table, N);
	
	hipFree(newer);
	hipFree(old);
	
	free(table);
	
	return(0);
}


void read_from_file(int *X, char *filename, int N){

  FILE *fp = fopen(filename, "r+");

  int size = fread(X, sizeof(int), N*N, fp);

  printf("elements: %d\n", size);

  fclose(fp);

}

void save_table(int *X, int N){

  FILE *fp;

  char filename[20];

  sprintf(filename, "cuda_c_table%dx%d.bin", N, N);

  printf("Saving table in file %s\n", filename);

  fp = fopen(filename, "w+");

  fwrite(X, sizeof(int), N*N, fp);

  fclose(fp);

}







